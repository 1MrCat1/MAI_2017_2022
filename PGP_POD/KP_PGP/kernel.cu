#include "hip/hip_runtime.h"


#include "hip/hip_runtime.h"
#include "PGP_KP_support_functions.h"
#include "time.h"
#include "mpi.h"
//�������� - ���������� ����������� ��������
//�������� ��� ���
//������� - 3� ���� aka �������

int LAUNCH_ARGS = 0;

__constant__ vec3 GPU_Lights[4];
__constant__ vec3 GPU_LightsColours[4];
__constant__ int GPU_LightsAmount;


__constant__ float GPU_AMBIENT = 0.3;
__constant__ float GPU_DIFFUSE = 0.55;
__constant__ float GPU_REFLECTION = 0.15;


__global__ void printmemorystate(uchar4* data,int sz) {
	if (threadIdx.x == 0) {
		printf("Memory State:\n");
		for (int i = 0; i < min(sz,100); ++i) {
			printf("%d: ", i);
			printUchar4(data[i]);
		}
	}
}

__global__ void printconstants() {
	if (threadIdx.x == 0) {
		printf("GPU_LightsAmount: %d\n", GPU_LightsAmount);
		for (int i = 0; i < GPU_LightsAmount; ++i) {
			printf("Light #%d\n", i);
			GPU_Lights[i].printToStdout();
			GPU_LightsColours[i].printToStdout();
		}
	}
}

texture<float4, 2, hipReadModeElementType> tex;

__global__ void SSAA(float4 *out, int w, int h, int resW, int resH) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int idy = blockDim.y * blockIdx.y + threadIdx.y;
	int offsetx = blockDim.x * gridDim.x;
	int offsety = blockDim.y * gridDim.y;
	int x, y;
	int multiplier = (w / resW) * (h / resH);
	float4 p;
	for (y = idy; y < h; y += offsety)
		for (x = idx; x < w; x += offsetx) {
			p = tex2D(tex, x, y);
			atomicAdd(&out[y / (h / resH) * resW + x / (w / resW)].x, p.x / multiplier);
			atomicAdd(&out[y / (h / resH) * resW + x / (w / resW)].y, p.y / multiplier);
			atomicAdd(&out[y / (h / resH) * resW + x / (w / resW)].z, p.z / multiplier);
		}
}

void CPU_SSAA(float4* in, float4* out, int w, int h, int resW, int resH) {
	int multiplier = (w / resW) * (h / resH);
	for (int y = 0; y < h; ++y) {
		for (int x = 0; x < w; ++x) {
			out[y / (h / resH) * resW + x / (w / resW)].x += in[y*w + x].x / multiplier;
			out[y / (h / resH) * resW + x / (w / resW)].y += in[y*w + x].y / multiplier;
			out[y / (h / resH) * resW + x / (w / resW)].z += in[y*w + x].z / multiplier;
		}
	}
}



__device__ vec3 GPUray(vec3 pos, vec3 dir, trig* trigs, int trigsAmount, int recursion) {
	int k, k_min = -1;
	vec3 e1, e2, p, t, q;
	float ts_min;
	for (k = 0; k < trigsAmount; k++) {
		e1 = diff(trigs[k].b, trigs[k].a);  
		e2 = diff(trigs[k].c, trigs[k].a); 
		p = prod(dir, e2);
		float div = dot(p, e1);
		if (fabs(div) < 1e-10)
			continue;
		t = diff(pos, trigs[k].a);
		float u = dot(p, t) / div;
		if (u < 0.0 || u > 1.0)
			continue;
		q = prod(t, e1);
		float v = dot(q, dir) / div;
		if (v < 0.0 || v + u > 1.0)
			continue;
		float ts = dot(q, e2) / div;
		if (ts < 0.0)
			continue;
		if (k_min == -1 || ts < ts_min) {
			k_min = k;
			ts_min = ts;
		}
	}
	if ((recursion == 1)&&(k_min==-1)) { 
		return {-1,-1,-1}; // �������� ����� �����
	}
	if (k_min == -1)
		return { 0, 0, 0};
	if (recursion > 0) {
		return trigs[k_min].color;
	}

	vec3 z = add(pos, multByNum(dir, ts_min - (float)(0.0001))); // ��-�� ������������ ����� ����������� ����� ���� ��� �����������
	e2 = diff(trigs[k_min].a, trigs[k_min].b);
	e1 = diff(trigs[k_min].b, trigs[k_min].c);
	vec3 resColour = multByNum(trigs[k_min].color, GPU_AMBIENT);
	p = norm(prod(e1, e2)); //������� ������������
	vec3 tmp;
	if (GPU_DIFFUSE > 1e-5) {
		for (int i = 0; i < GPU_LightsAmount; ++i) {
			t = norm(diff(GPU_Lights[i], z)); //������. ������� � ��������� �����
			if (vec3Eq(GPUray(z, t, trigs, trigsAmount, 1), { -1,-1,-1 })) {
				float r = modulus(GPU_Lights[i], z); // ���������� �� �����
				//printf("Diffuse light in effect\n");
				tmp = multByNum(GPU_LightsColours[i], GPU_DIFFUSE*min(1, LIGHT_FADING_RADIUS / max(r, 0.000001)));
				resColour = addlimited(resColour, tmp);
			}
		}
	}
	if (GPU_REFLECTION > 1e-5) {
		if (dot(p, dir) > 0) {
			p = multByNum(p, -1);
		}

		tmp = GPUray(z, diff(dir, multByNum(multByNum(p, dot(dir, p)), 2)), trigs, trigsAmount, 2);
		if (abs(trigs[k_min].reflect) > EQ_EPSILON) {

			tmp = multByNum(tmp, GPU_REFLECTION*trigs[k_min].reflect);
			resColour = addlimited(resColour, tmp);

		}
	}
	return resColour;
}

__global__ void GPURender(vec3 Camera, vec3 bx, vec3 by, vec3 bz, int w, int h, float dw, float dh, float z, vec3* data, trig* trigs, int trigsAmount) {
	int idx = blockDim.x*blockIdx.x + threadIdx.x;
	int idy = blockDim.y*blockIdx.y + threadIdx.y;
	int offsetx = blockDim.x*gridDim.x;
	int offsety = blockDim.y*gridDim.y;
	for (int j = idy; j < h; j+=offsety) {
		for (int i = idx; i < w; i+=offsetx) {
			vec3 v = { (float)(-1.0) + dw * i, ((float)(-1.0) + dh * j) * h / w, z };
			//����������� ������� ���� � ����� ������
			vec3 dir = mult(bx, by,bz, v);
			data[(h - 1 - j) * w + i] = GPUray(Camera, norm(dir), trigs, trigsAmount,0);
		}
	}
}


int main(int argc, char* argv[]) {
	/*vec3 test;
	test.x = 1;
	test.y = 2;
	test.z = 3;
	printf("Size: %d Addresses:%X %X %X %X", sizeof(test), &test, &test.x, &test.y, &test.z);
	return 0;*/
	int frames, frameH, frameW, frameAngle;
	char outputPath[256];

	float r0c, z0c, phi0c, Arc, Azc, omegarc, omegazc, omegaphic, prc, pzc;
	float r0n, z0n, phi0n, Arn, Azn, omegarn, omegazn, omegaphin, prn, pzn;

	vec3 center1, center2, center3;
	vec3 colour1, colour2, colour3;
	float radius1, radius2, radius3;
	float reflect1, reflect2, reflect3;
	float transp1, transp2, transp3;
	float lights1, lights2, lights3;

	vec3 floor[4];
	char floorTexturePath[128];
	vec3 floorColour;
	float floorReflect;


	int maxRecursionDepth;
	int raysPerPixelSqrt;

	int i, j;// , k;

	int numproc, id;
	MPI_Status status;
	MPI_Init(&argc, &argv);
	MPI_Comm_size(MPI_COMM_WORLD, &numproc);
	MPI_Comm_rank(MPI_COMM_WORLD, &id);

	MPI_Barrier(MPI_COMM_WORLD);
	//printf("ID: %d, TOTAL: %d\n", id, numproc);

	if (id == 0) {

		if (ProcessInputArguments(argc, argv, &LAUNCH_ARGS)) {
			fprintf(stderr, "Encountered error while processing launch aruments. Exiting...\n");
			exit(0);
		};
		if (LAUNCH_ARGS&DEFAULT) {
			PrintKrasivoe();
			return 0;
		}

		//printf("%d\n", LAUNCH_ARGS);
		//FILE* inputFile = stdin;
		FILE* inputFile = fopen("KP_input.txt", "r");
		//Data input
		fscanf(inputFile, "%d", &frames);
		fscanf(inputFile, "%s", outputPath);
		fscanf(inputFile, "%d %d %d", &frameW, &frameH, &frameAngle);
		fscanf(inputFile, "%f %f %f %f %f %f %f %f %f %f", &r0c, &z0c, &phi0c, &Arc, &Azc, &omegarc, &omegazc, &omegaphic, &prc, &pzc);
		fscanf(inputFile, "%f %f %f %f %f %f %f %f %f %f", &r0n, &z0n, &phi0n, &Arn, &Azn, &omegarn, &omegazn, &omegaphin, &prn, &pzn);

		center1.readFromStream(inputFile);
		colour1.readFromStream(inputFile);
		fscanf(inputFile, "%f %f %f %f", &radius1, &reflect1, &transp1, &lights1);


		center2.readFromStream(inputFile);
		colour2.readFromStream(inputFile);
		fscanf(inputFile, "%f %f %f %f", &radius2, &reflect2, &transp2, &lights2);

		center3.readFromStream(inputFile);
		colour3.readFromStream(inputFile);
		fscanf(inputFile, "%f %f %f %f", &radius3, &reflect3, &transp3, &lights3);

		for (i = 0; i < 4; ++i) {
			floor[i].readFromStream(inputFile);
		}
		fscanf(inputFile, "%s", floorTexturePath);
		floorColour.readFromStream(inputFile);
		fscanf(inputFile, "%f", &floorReflect);

		lightsAmount = (int*)malloc(sizeof(int));
		fscanf(inputFile, "%d", lightsAmount);

		Lights = (vec3*)malloc(sizeof(vec3)*(*lightsAmount));
		LightsColours = (vec3*)malloc(sizeof(vec3)*(*lightsAmount));

		for (i = 0; i < *lightsAmount; ++i) {
			Lights[i].readFromStream(inputFile);
			LightsColours[i].readFromStream(inputFile);
		}
		fscanf(inputFile, "%d %d", &maxRecursionDepth, &raysPerPixelSqrt);
	}

	MPI_Bcast(&LAUNCH_ARGS, 1, MPI_INT, 0, MPI_COMM_WORLD);

	MPI_Bcast(&frames, 1, MPI_INT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&frameW, 1, MPI_INT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&frameH, 1, MPI_INT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&frameAngle, 1, MPI_INT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&raysPerPixelSqrt, 1, MPI_INT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&maxRecursionDepth, 1, MPI_INT, 0, MPI_COMM_WORLD);

	MPI_Bcast(outputPath, 256, MPI_CHAR, 0, MPI_COMM_WORLD);
	MPI_Bcast(floorTexturePath, 128, MPI_CHAR, 0, MPI_COMM_WORLD);

	MPI_Bcast(&r0c, 1, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&z0c, 1, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&phi0c, 1, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&Arc, 1, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&Azc, 1, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&omegarc, 1, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&omegazc, 1, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&omegaphic, 1, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&prc, 1, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&pzc, 1, MPI_FLOAT, 0, MPI_COMM_WORLD);
	

	MPI_Bcast(&r0n, 1, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&z0n, 1, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&phi0n, 1, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&Arn, 1, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&Azn, 1, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&omegarn, 1, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&omegazn, 1, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&omegaphin, 1, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&prn, 1, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&pzn, 1, MPI_FLOAT, 0, MPI_COMM_WORLD);


	MPI_Bcast(&radius1, 1, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&radius2, 1, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&radius3, 1, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&reflect1, 1, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&reflect2, 1, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&reflect3, 1, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&transp1, 1, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&transp2, 1, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&transp3, 1, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&lights1, 1, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&lights2, 1, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&lights3, 1, MPI_FLOAT, 0, MPI_COMM_WORLD);

	MPI_Bcast(&floorReflect, 1, MPI_FLOAT, 0, MPI_COMM_WORLD);


	MPI_Bcast(&center1, 3, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&center2, 3, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&center3, 3, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&colour1, 3, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&colour2, 3, MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&colour3, 3, MPI_FLOAT, 0, MPI_COMM_WORLD);

	//for (i = 0; i < 4; ++i) {
	MPI_Bcast(floor, 12, MPI_FLOAT, 0, MPI_COMM_WORLD);
	//}

	MPI_Bcast(&floorColour, 3, MPI_FLOAT, 0, MPI_COMM_WORLD);

	if (id != 0) {
		lightsAmount = (int*)malloc(sizeof(int));
	}

	MPI_Bcast(lightsAmount, 1, MPI_INT, 0, MPI_COMM_WORLD);

	if (id != 0) {
		Lights = (vec3*)malloc(sizeof(vec3)*(*lightsAmount));
		LightsColours = (vec3*)malloc(sizeof(vec3)*(*lightsAmount));
	}
	
	MPI_Bcast(Lights, 3* (*lightsAmount), MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Bcast(LightsColours, 3 * (*lightsAmount), MPI_FLOAT, 0, MPI_COMM_WORLD);
	MPI_Barrier(MPI_COMM_WORLD);


	int framesPerProccess = frames / numproc;
	int framesLeft = frames % numproc;
	int startingFrame = framesPerProccess * id;
	startingFrame += (id >= framesLeft ? framesLeft : id);
	int endingFrame = startingFrame + framesPerProccess + (id >= framesLeft ? 0 : 1);
	//printf("FPP: %d, FL:%d Start:%d Last:%d\n", framesPerProccess, framesLeft,startingFrame,endingFrame);
	MPI_Barrier(MPI_COMM_WORLD);

	//---------------------------------------------------------------------------------------------------

	clock_t start, end;
	
	int renderH = frameH * raysPerPixelSqrt;
	int renderW = frameW * raysPerPixelSqrt;
	int trigsAmount = 2 + 4 + 12 + 8;
	trig* trigs = BuildSpace(floor,floorColour,floorReflect,center1,colour1,radius1,reflect1, center2, colour2, radius2, reflect2, center3, colour3, radius3,reflect3);
	//������ ���� �� ������ ���
	float t;
	vec3 cameraPoint, cameraDirection;
	char outputBuff[264];
	if (frames > 1e6) {
		fprintf(stderr, "ERROR: string buffer for output file name is too small :(\nReduce number of frames please\n");
		free(Lights);
		free(LightsColours);
		return 0;
	}


	vec3* pixels;// = (vec3*)malloc(sizeof(vec3)*renderW*renderH);
	float4* f4pixels;// = (float4*)malloc(sizeof(float4)*renderW*renderH);
	uchar4* ssaaPixels;// = (uchar4*)malloc(sizeof(uchar4)*frameW*frameH);
	float4* floatPixels;// = (float4*)malloc(sizeof(float4)*frameW*frameH);

	float lightDz = (float)1/frames;
	
	LightsColours[1].y -= lightDz * startingFrame;
	LightsColours[1].z -= lightDz * startingFrame;

	hipEvent_t Gstart, Gend;
	float gputime;

	vec3* GPUpixels;
	float4* GPUfloatpixels;
	trig* GPUtrigs;
	hipArray *arr;
	hipChannelFormatDesc ch = hipCreateChannelDesc<float4>();

	int deviceCount;
	CSC(hipGetDeviceCount(&deviceCount));
	if (deviceCount != 1) {
		CSC(hipSetDevice(id & deviceCount));
	}
	MPI_Barrier(MPI_COMM_WORLD);
	if (LAUNCH_ARGS&GPU) {

		CSC(hipMalloc(&GPUpixels, sizeof(vec3)*renderW*renderH));
		CSC(hipMalloc(&GPUtrigs, sizeof(trig)*trigsAmount));
		CSC(hipMemcpy(GPUtrigs, trigs, sizeof(trig)*trigsAmount, hipMemcpyHostToDevice));
		//CSC(hipMemcpy(GPUpixels, pixels, sizeof(vec3)*renderW*renderH, hipMemcpyHostToDevice));
		
		CSC(hipEventCreate(&Gstart));
		CSC(hipEventCreate(&Gend));
		
		CSC(hipMallocArray(&arr, &ch, renderW, renderH));
		
		tex.addressMode[0] = hipAddressModeClamp;
		tex.addressMode[1] = hipAddressModeClamp;
		tex.channelDesc = ch;
		tex.filterMode = hipFilterModePoint;
		tex.normalized = false;
		
		CSC(hipBindTextureToArray(tex, arr, ch));
		
		CSC(hipMalloc(&GPUfloatpixels, sizeof(float4)*frameW*frameH));
	

		CSC(hipMemcpyToSymbol(HIP_SYMBOL(GPU_Lights), Lights, sizeof(vec3) * (*lightsAmount), 0, hipMemcpyHostToDevice));
		CSC(hipMemcpyToSymbol(HIP_SYMBOL(GPU_LightsColours), LightsColours, sizeof(vec3) * (*lightsAmount), 0, hipMemcpyHostToDevice));
		CSC(hipMemcpyToSymbol(HIP_SYMBOL(GPU_LightsAmount), lightsAmount, sizeof(int), 0, hipMemcpyHostToDevice));

	}

	MPI_Barrier(MPI_COMM_WORLD);
	vec3 bx, by, bz;
	float dw, dh, z;
	clock_t globStart, globEnd;
	globStart = clock();
	if (LAUNCH_ARGS&CPU) {
		#pragma omp parallel for private(i,t,cameraPoint,cameraDirection,pixels,f4pixels,ssaaPixels,floatPixels) schedule(dynamic)
		for (int i = startingFrame; i < endingFrame; ++i) {
			//printf("ID:%d,FRAME:%d\n", id, i);
			pixels = (vec3*)malloc(sizeof(vec3)*renderW*renderH);
			f4pixels = (float4*)malloc(sizeof(float4)*renderW*renderH);
			ssaaPixels = (uchar4*)malloc(sizeof(uchar4)*frameW*frameH);
			floatPixels = (float4*)malloc(sizeof(float4)*frameW*frameH);
			t = i * (float)(2 * M_PI / frames);

			cameraPoint = { (r0c + Arc * sin((omegarc*t + prc)))*cos((phi0c + omegaphic * t)),
							(r0c + Arc * sin((omegarc*t + prc)))*sin((phi0c + omegaphic * t)),
							z0c + Azc * sin((omegazc*t + pzc))
			};
			cameraDirection = { (r0n + Arn * sin((omegarn*t + prn)))*cos((phi0n + omegaphin * t)),
								(r0n + Arn * sin((omegarn*t + prn)))*sin((phi0n + omegaphin * t)),
								z0n + Azn * sin((omegazn*t + pzn))
			};
			//printf("Frame: %d\n", i);


			//printf("Running CPU render %d\n", i);
			start = clock();
			render(cameraPoint, cameraDirection, renderW, renderH, (float)frameAngle, pixels, trigs, 2 + 4 + 12 + 8);
			end = clock();
			printf("ID: %d Frame: %d CPU: %ld ms\n", id, i, end - start);

			convertVec3ToFloat4(pixels, f4pixels, renderW*renderH);

			for (j = 0; j < frameW*frameH; ++j) {
				floatPixels[j] = { 0,0,0,0 };
			}

			//printf("Starting CPU SSAA %d\n", i);
			start = clock();
			CPU_SSAA(f4pixels, floatPixels, renderW, renderH, frameW, frameH);
			end = clock();
			printf("ID: %d Frame: %d CPU SSAA: %ld ms\n", id, i, end - start);


			for (j = 0; j < frameW*frameH; ++j) {
				ssaaPixels[j] = UcharFromNormalFloat4(floatPixels[j]);
			}

			sprintf(outputBuff, outputPath, i);
			printf("ID: %d Frame: %d - %s\n", id, i, outputBuff);

			FILE *out = fopen(outputBuff, "wb");
			if (out == NULL) {
				fprintf(stderr, "ERROR: Can't create output file :(\n");
				break;
			}
			fwrite(&frameW, sizeof(int), 1, out);
			fwrite(&frameH, sizeof(int), 1, out);
			fwrite(ssaaPixels, sizeof(uchar4), frameW * frameH, out);
			fclose(out);

			free(pixels);
			free(f4pixels);
			free(ssaaPixels);
			free(floatPixels);
		}
	}
	if (LAUNCH_ARGS&GPU) {
		pixels = (vec3*)malloc(sizeof(vec3)*renderW*renderH);
		f4pixels = (float4*)malloc(sizeof(float4)*renderW*renderH);
		ssaaPixels = (uchar4*)malloc(sizeof(uchar4)*frameW*frameH);
		floatPixels = (float4*)malloc(sizeof(float4)*frameW*frameH);
		for (int i = startingFrame; i < endingFrame; ++i) {
			//printf("ID:%d,FRAME:%d\n", id, i);
			t = i * (float)(2 * M_PI / frames);

			cameraPoint = { (r0c + Arc * sin((omegarc*t + prc)))*cos((phi0c + omegaphic * t)),
							(r0c + Arc * sin((omegarc*t + prc)))*sin((phi0c + omegaphic * t)),
							z0c + Azc * sin((omegazc*t + pzc))
			};
			cameraDirection = { (r0n + Arn * sin((omegarn*t + prn)))*cos((phi0n + omegaphin * t)),
								(r0n + Arn * sin((omegarn*t + prn)))*sin((phi0n + omegaphin * t)),
								z0n + Azn * sin((omegazn*t + pzn))
			};
			//printf("Frame: %d\n", i);

			dw = (float)(2.0 / (renderW - 1.0));
			dh = (float)(2.0 / (renderH - 1.0));
			z = (float)(1.0 / tan(frameAngle * M_PI / 360.0));
			//����� � ����� ������
			bz = norm(diff(cameraDirection, cameraPoint)); // ����������� ������
			bx = norm(prod(bz, { 0.0, 0.0, 1.0 })); // � ����
			by = norm(prod(bx, bz)); // �������

			CSC(hipEventRecord(Gstart));

			GPURender << <dim3(256, 256), dim3(16, 16) >> > (cameraPoint, bx, by, bz, renderW, renderH, dw, dh, z, GPUpixels, GPUtrigs, trigsAmount);

			CSC(hipGetLastError());
			//printmemorystate << <1, 1 >> > (GPUpixels, renderH*renderW);
			CSC(hipGetLastError());
			CSC(hipEventRecord(Gend));
			CSC(hipEventSynchronize(Gend));
			CSC(hipEventElapsedTime(&gputime, Gstart, Gend));
			printf("ID: %d Frame: %d GPU: %f ms\n",id,i, gputime); //end - start);
			CSC(hipMemcpy(pixels, GPUpixels, sizeof(vec3)*renderW*renderH, hipMemcpyDeviceToHost));

			convertVec3ToFloat4(pixels, f4pixels, renderW*renderH);

			for (j = 0; j < frameW*frameH; ++j) {
				floatPixels[j] = { 0,0,0,0 };
			}

			CSC(hipEventRecord(Gstart));

			CSC(hipMemcpyToArray(arr, 0, 0, f4pixels, sizeof(float4) * renderW * renderH, hipMemcpyHostToDevice));
			CSC(hipMemcpy(GPUfloatpixels, floatPixels, sizeof(float4)*frameW*frameH, hipMemcpyHostToDevice));

			SSAA << <dim3(32, 32), dim3(16, 16) >> > (GPUfloatpixels, renderW, renderH, frameW, frameH);

			CSC(hipMemcpy(floatPixels, GPUfloatpixels, sizeof(float4)*frameW*frameH, hipMemcpyDeviceToHost));

			CSC(hipGetLastError());
			CSC(hipEventRecord(Gend));
			CSC(hipEventSynchronize(Gend));
			CSC(hipEventElapsedTime(&gputime, Gstart, Gend)); 
			printf("ID: %d Frame: %d SSAA+memcpy: %f ms\n", id, i, gputime);


			for (j = 0; j < frameW*frameH; ++j) {
				ssaaPixels[j] = UcharFromNormalFloat4(floatPixels[j]);
			}
			sprintf(outputBuff, outputPath, i);
			printf("ID: %d Frame: %d - %s\n", id, i, outputBuff);

			FILE *out = fopen(outputBuff, "wb");
			if (out == NULL) {
				fprintf(stderr, "ERROR: Can't create output file :(\n");
				break;
			}
			fwrite(&frameW, sizeof(int), 1, out);
			fwrite(&frameH, sizeof(int), 1, out);
			fwrite(ssaaPixels, sizeof(uchar4), frameW * frameH, out);
			fclose(out);

			LightsColours[1].y -= lightDz;
			LightsColours[1].z -= lightDz;
			//LightsColours[1].printToStdout();
			CSC(hipMemcpyToSymbol(HIP_SYMBOL(GPU_LightsColours), LightsColours, sizeof(vec3) * (*lightsAmount), 0, hipMemcpyHostToDevice));

		}
		free(pixels);
		free(f4pixels);
		free(ssaaPixels);
		free(floatPixels);
	}

	globEnd = clock();
	printf("ID: %d. Summary: %ld ms",id, globEnd - globStart);
	free(trigs);
	free(Lights);
	free(LightsColours);
	if (LAUNCH_ARGS&GPU) {
		CSC(hipUnbindTexture(tex));
		CSC(hipFreeArray(arr));
		CSC(hipFree(GPUfloatpixels));
		CSC(hipFree(GPUpixels));
		CSC(hipFree(GPUtrigs));
		//CSC(hipFree(GPUfloatpixels));
	}
	MPI_Finalize();
	//CSC(hipFree(GPUtrigs));
	//CSC(hipFree(GPUpixels));
	return 0;
}
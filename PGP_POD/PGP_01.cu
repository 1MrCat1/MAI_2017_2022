#include "hip/hip_runtime.h"
#include "cuda_device_runtime_api.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define CSC(call)  													\
do {																\
	hipError_t res = call;											\
	if (res != hipSuccess) {										\
		fprintf(stderr, "ERROR in %s:%d. Message: %s\n",			\
				__FILE__, __LINE__, hipGetErrorString(res));		\
		exit(0);													\
	}																\
} while(0)

__global__ void Kernel(double* a, double* b, int sz)
{
	int idx = (blockDim.x)*(blockIdx.x) + threadIdx.x;
	int offset = blockDim.x*gridDim.x;
	for (int i = idx; i < sz; i += offset) {
		a[i] = fmax(a[i],b[i]);

	}
}


int main() {
	int i, sz;
	scanf("%d", &sz);
	double* a = (double*)malloc(sizeof(double)*sz);
	double* b = (double*)malloc(sizeof(double)*sz);
	double* dev_a;
	double* dev_b;
	CSC(hipMalloc(&dev_a, sizeof(double)*sz));
	CSC(hipMalloc(&dev_b, sizeof(double)*sz));
	for (i = 0; i < sz; ++i) {
		scanf("%lf", &a[i]);
	}
	for (i = 0; i < sz; ++i) {
		scanf("%lf", &b[i]);
	}
	CSC(hipMemcpy(dev_a, a, sizeof(double)*sz, hipMemcpyHostToDevice));
	CSC(hipMemcpy(dev_b, b, sizeof(double)*sz, hipMemcpyHostToDevice));
	
	Kernel <<<256, 256 >>> (dev_a, dev_b, sz);

	CSC(hipMemcpy(a, dev_a, sizeof(double)*sz, hipMemcpyDeviceToHost));
	for (i = 0; i < sz; ++i) {
		printf("%.10e\n", a[i]);
	}
	free(a);
	free(b);
	hipFree(dev_a);
	hipFree(dev_b);
	return 0;
}
